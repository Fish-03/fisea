#include "hip/hip_runtime.h"
#include "testfn.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}

void call_hello_from_gpu() {
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();
}