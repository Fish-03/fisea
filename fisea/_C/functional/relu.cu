#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h> // Why this is not neccessary?
// #include <>
#include <iostream>

#include "../handler.cuh"

__global__ void relu_gpu(float* in, float* out, int n) {
  CUDA_KERNEL_LOOP(i, n) {
  out[i] = in[i] > 0 ? in[i] : 0;
  }
}
