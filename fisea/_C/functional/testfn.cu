#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h> // Why this is not neccessary?
// #include <>
#include <iostream>

#include "testfn.cuh"

__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}

void call_hello_from_gpu() {
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();
}